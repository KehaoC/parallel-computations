#include "utils.h"
#include <hipDNN.h>

// cuDNN错误检查宏
#define CHECK_CUDNN_ERROR(call) \
do { \
    hipdnnStatus_t err = call; \
    if (err != HIPDNN_STATUS_SUCCESS) { \
        fprintf(stderr, "cuDNN Error in %s:%d: %s\n", __FILE__, __LINE__, \
                hipdnnGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// cuDNN卷积实现
void cudnn_conv(float *h_input, float *h_kernel, float *h_output,
               int in_height, int in_width, int kernel_size, int stride, int padding, int channels,
               double *time_taken) {
    // 计算输出尺寸
    int out_height = (in_height + 2 * padding - kernel_size) / stride + 1;
    int out_width = (in_width + 2 * padding - kernel_size) / stride + 1;
    
    // 计算内存大小
    size_t input_size = channels * in_height * in_width * sizeof(float);
    size_t kernel_size_bytes = channels * channels * kernel_size * kernel_size * sizeof(float);
    size_t output_size = channels * out_height * out_width * sizeof(float);
    
    // 分配设备内存
    float *d_input, *d_kernel, *d_output;
    CHECK_CUDA_ERROR(hipMalloc(&d_input, input_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_kernel, kernel_size_bytes));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, output_size));
    
    // 将数据从主机复制到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_kernel, h_kernel, kernel_size_bytes, hipMemcpyHostToDevice));
    
    // 创建cuDNN句柄
    hipdnnHandle_t cudnn;
    CHECK_CUDNN_ERROR(hipdnnCreate(&cudnn));
    
    // 创建张量描述符
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    
    CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN_ERROR(hipdnnCreateFilterDescriptor(&kernel_desc));
    CHECK_CUDNN_ERROR(hipdnnCreateConvolutionDescriptor(&conv_desc));
    
    // 设置张量描述符
    // NCHW格式：批次大小(N)=1, 通道数(C)=channels, 高度(H)=in_height, 宽度(W)=in_width
    CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                               1, channels, in_height, in_width));
    CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                               1, channels, out_height, out_width));
    
    // 设置卷积核描述符
    // KCRS格式：输出通道数(K)=channels, 输入通道数(C)=channels, 高度(R)=kernel_size, 宽度(S)=kernel_size
    CHECK_CUDNN_ERROR(hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                               channels, channels, kernel_size, kernel_size));
    
    // 设置卷积描述符
    CHECK_CUDNN_ERROR(hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride,
                                                    1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    
    // 获取最优算法
    hipdnnConvolutionFwdAlgo_t algo;
    int returnedAlgoCount;
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    CHECK_CUDNN_ERROR(hipdnnFindConvolutionForwardAlgorithm(cudnn, input_desc, kernel_desc, conv_desc, output_desc,
                                                        1, &returnedAlgoCount, &perfResults));
    algo = perfResults.algo;
    
    // 获取工作空间大小
    size_t workspace_size = 0;
    CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, kernel_desc, conv_desc, output_desc,
                                                            algo, &workspace_size));
    
    // 分配工作空间
    void *workspace = nullptr;
    if (workspace_size > 0) {
        CHECK_CUDA_ERROR(hipMalloc(&workspace, workspace_size));
    }
    
    // 执行卷积并计时
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    hipDeviceSynchronize();
    double start_time = get_time();
    
    CHECK_CUDNN_ERROR(hipdnnConvolutionForward(cudnn, &alpha, input_desc, d_input, kernel_desc, d_kernel,
                                            conv_desc, algo, workspace, workspace_size, &beta,
                                            output_desc, d_output));
    
    hipDeviceSynchronize();
    double end_time = get_time();
    *time_taken = end_time - start_time;
    
    // 将结果从设备复制到主机
    CHECK_CUDA_ERROR(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost));
    
    // 清理资源
    if (workspace) {
        CHECK_CUDA_ERROR(hipFree(workspace));
    }
    CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
    CHECK_CUDNN_ERROR(hipdnnDestroyFilterDescriptor(kernel_desc));
    CHECK_CUDNN_ERROR(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN_ERROR(hipdnnDestroy(cudnn));
    
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_kernel));
    CHECK_CUDA_ERROR(hipFree(d_output));
} 